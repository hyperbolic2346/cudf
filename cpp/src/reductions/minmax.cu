#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
// The translation unit for reduction `minmax`

#include <thrust/transform_reduce.h>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/utilities/device_operators.cuh>
#include <cudf/reduction.hpp>
#include <cudf/scalar/scalar_factories.hpp>
#include <type_traits>

namespace cudf {
namespace detail {

namespace {
/**
 * @brief stores the minimum and maximum
 * values that have been encountered so far
 *
 */
template <typename T>
struct minmax_pair {
  T min_val;
  T max_val;
  bool min_valid;
  bool max_valid;

  __host__ __device__ minmax_pair()
    : min_val(cudf::DeviceMin::identity<T>()),
      max_val(cudf::DeviceMax::identity<T>()),
      min_valid(false),
      max_valid(false){};
  __host__ __device__ minmax_pair(T val, bool valid_)
    : min_val(val), max_val(val), min_valid(valid_), max_valid(valid_){};
  __host__ __device__ minmax_pair(T min_val_, bool min_valid_, T max_val_, bool max_valid_)
    : min_val(min_val_), max_val(max_val_), min_valid(min_valid_), max_valid(max_valid_){};
};

/**
 * @brief functor that accepts two minmax_pairs and returns a
 * minmax_pair whose minimum and maximum values are the min() and max()
 * respectively of the minimums and maximums of the input pairs. Respects
 * validity.
 *
 */
template <typename T>
struct minmax_with_null_binary_op
  : public thrust::binary_function<minmax_pair<T>, minmax_pair<T>, minmax_pair<T>> {
  __host__ __device__ minmax_pair<T> operator()(const minmax_pair<T> &x,
                                                const minmax_pair<T> &y) const
  {
    T x_min = x.min_valid ? x.min_val : cudf::DeviceMin::identity<T>();
    T y_min = y.min_valid ? y.min_val : cudf::DeviceMin::identity<T>();
    T x_max = x.max_valid ? x.max_val : cudf::DeviceMax::identity<T>();
    T y_max = y.max_valid ? y.max_val : cudf::DeviceMax::identity<T>();

    // The only invalid situation is if we compare two invalid values.
    // Otherwise, we are certain to select a valid value due to the
    // identity functions above changing the comparison value.
    bool valid_min_result = x.min_valid || y.min_valid;
    bool valid_max_result = x.max_valid || y.max_valid;

    return minmax_pair<T>{
      thrust::min(x_min, y_min), valid_min_result, thrust::max(x_max, y_max), valid_max_result};
  }
};

/**
 * @brief functor that accepts two minmax_pairs and returns a
 * minmax_pair whose minimum and maximum values are the min() and max()
 * respectively of the minimums and maximums of the input pairs. Expects
 * no null values.
 *
 */
template <typename T>
struct minmax_no_null_binary_op
  : public thrust::binary_function<minmax_pair<T>, minmax_pair<T>, minmax_pair<T>> {
  __host__ __device__ minmax_pair<T> operator()(const minmax_pair<T> &x,
                                                const minmax_pair<T> &y) const
  {
    return minmax_pair<T>{
      thrust::min(x.min_val, y.min_val), true, thrust::max(x.max_val, y.max_val), true};
  }
};

/**
 * @brief functor that calls thrust::transform_reduce to produce a std::pair
 * of scalars that represent the minimum and maximum values of the input data
 * respectively. Note that dictionaries and non-relationally comparable objects
 * are not supported.
 *
 */
struct minmax_functor {
  template <typename T>
  // unable to support fixed point due to DeviceMin/DeviceMax not supporting fixed point
  std::enable_if_t<cudf::is_relationally_comparable<T, T>() and
                     not std::is_same<T, dictionary32>::value and not cudf::is_fixed_point<T>(),
                   std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>>>
  operator()(const cudf::column_view &col, rmm::mr::device_memory_resource *mr, hipStream_t stream)
  {
    auto device_col = column_device_view::create(col, stream);

    // compute minimum and maximum values
    minmax_pair<T> result;
    if (col.nullable()) {
      result = thrust::transform_reduce(
        thrust::make_counting_iterator<size_type>(0),
        thrust::make_counting_iterator<size_type>(col.size()),
        [d_col = *device_col] __device__(size_type index) -> minmax_pair<T> {
          return minmax_pair<T>(d_col.element<T>(index), d_col.is_valid(index));
        },
        minmax_pair<T>{},
        minmax_with_null_binary_op<T>{});
    } else {
      result = thrust::transform_reduce(
        thrust::make_counting_iterator<size_type>(0),
        thrust::make_counting_iterator<size_type>(col.size()),
        [d_col = *device_col] __device__(size_type index) -> minmax_pair<T> {
          return minmax_pair<T>(d_col.element<T>(index), d_col.is_valid(index));
        },
        minmax_pair<T>{},
        minmax_no_null_binary_op<T>{});
    }

    std::unique_ptr<scalar> min =
      make_fixed_width_scalar<T>(result.min_val, result.min_valid, stream, mr);
    std::unique_ptr<scalar> max =
      make_fixed_width_scalar<T>(result.max_val, result.max_valid, stream, mr);
    return {std::move(min), std::move(max)};
  }

  template <typename T,
            std::enable_if_t<not cudf::is_relationally_comparable<T, T>() or
                             cudf::is_fixed_point<T>()> * = nullptr>
  std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> operator()(
    const cudf::column_view &col, rmm::mr::device_memory_resource *mr, hipStream_t stream)
  {
    CUDF_FAIL("type not supported");
  }

  template <typename T, typename std::enable_if_t<std::is_same<T, dictionary32>::value> * = nullptr>
  std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> operator()(
    const cudf::column_view &col, rmm::mr::device_memory_resource *mr, hipStream_t stream)
  {
    CUDF_FAIL("dictionary type not supported");
  }
};

}  // namespace

std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> minmax(
  const cudf::column_view &col,
  rmm::mr::device_memory_resource *mr = rmm::mr::get_current_device_resource(),
  hipStream_t stream                 = 0)
{
  return type_dispatcher(col.type(), minmax_functor{}, col, mr, stream);
}
}  // namespace detail

/**
 * @copydoc cudf::minmax
 */
std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> minmax(
  const cudf::column_view &col, rmm::mr::device_memory_resource *mr)
{
  return cudf::detail::minmax(col, mr);
}

}  // namespace cudf
